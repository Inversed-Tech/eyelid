#include "hip/hip_runtime.h"
#include "fq79_2x64.cuh"

// Endianness checker in case we reinterpret Rust data with different integer types.
extern "C" __global__ void endianness_check(const u128 *x, u64 *out_low, u64 *out_high, u64 *out_one)
{
  *out_low = (u64)*x;
  *out_high = (u64)(*x >> 64);
  *out_one = 1;
}
